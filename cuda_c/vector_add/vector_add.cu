
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000000

//GPU Kernel
__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 

    // Allocates memory and returns a pointer to it.
    a   = (float*)malloc(sizeof(float) * N); // size in bytes
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Main function
    vector_add<<<1,1>>>(out, a, b, N);
}
